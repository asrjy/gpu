#include "hip/hip_runtime.h"
#include <iostream>
#include <vector>
#include <hip/hip_runtime.h>
#include "helpers.h"

__global__
void divergent_kernel(int* data, int n){
    // kernel with warp divergence
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < n){
        if (idx % 2 == 0){
            data[idx] *= 2;
        } else {
            data[idx] += 1;
        }
    }
}

__global__
void optimized_kernel(int* data, int n){
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if(idx < n){
        // process indices as well
        if (idx%2 == 0){
            data[idx] *= 2;
        }
        // synchronize threads within warp
        __syncwarp();
        if (idx%2 != 0){
            data[idx] += 1;
        }
    }
}

void initialize_data(std::vector<int>& data){
    for(int i = 0; i < data.size(); ++i){
        data[i] = i;
    }
}

bool verify_results(const std::vector<int>& data){
    for (int i =0; i < data.size(); i++){
        int expected = (i % 2 == 0) ? i * 2: i+1;
        if(data[i] != expected){
            std::cout << "mismatch at " << i << ": " << data[i] << " != " << expected << std::endl;
            return false;
        }
    }
    return true; 
}

int main(){
    const int N = 1 << 24;
    const int BLOCK_SIZE = 256;

    std::vector<int> h_data(N);
    initialize_data(h_data);

    int *d_data;
    CUDA_CHECK(hipMalloc(&d_data, N * sizeof(int)));

    hipEvent_t start, stop;
    CUDA_CHECK(hipEventCreate(&start));
    CUDA_CHECK(hipEventCreate(&stop));

    CUDA_CHECK(hipMemcpy(d_data, h_data.data(), N * sizeof(int), hipMemcpyHostToDevice));

    CUDA_CHECK(hipEventRecord(start));
    divergent_kernel<<<(N + BLOCK_SIZE - 1)/BLOCK_SIZE, BLOCK_SIZE>>>(d_data, N);
    CUDA_CHECK(hipEventRecord(stop));

    CUDA_CHECK(hipMemcpy(h_data.data(), d_data, N * sizeof(int), hipMemcpyDeviceToHost));

    float divergent_ms = 0;
    CUDA_CHECK(hipEventElapsedTime(&divergent_ms, start, stop));

    bool divergent_correct = verify_results(h_data);

    initialize_data(h_data);
    CUDA_CHECK(hipMemcpy(d_data, h_data.data(), N * sizeof(int), hipMemcpyHostToDevice));

    CUDA_CHECK(hipEventRecord(start));
    optimized_kernel<<<(N + BLOCK_SIZE - 1) / BLOCK_SIZE, BLOCK_SIZE>>>(d_data, N);
    CUDA_CHECK(hipEventRecord(stop));
    
    CUDA_CHECK(hipMemcpy(h_data.data(), d_data, N * sizeof(int), hipMemcpyDeviceToHost));
    
    float optimized_ms = 0;
    CUDA_CHECK(hipEventElapsedTime(&optimized_ms, start, stop));
    
    bool optimized_correct = verify_results(h_data);

    std::cout << "divergent kernel time: " << divergent_ms << " ms" << std::endl;
    std::cout << "optimized kernel time: " << optimized_ms << " ms" << std::endl;
    std::cout << "speedup: " << divergent_ms / optimized_ms << "x" << std::endl;

    CUDA_CHECK(hipFree(d_data));
    CUDA_CHECK(hipEventDestroy(start));
    CUDA_CHECK(hipEventDestroy(stop));

    return 0;

}